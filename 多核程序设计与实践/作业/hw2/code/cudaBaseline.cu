#include "hip/hip_runtime.h"
//#include<hip/hip_fp16.h>
#include <stdio.h>
#include <stdlib.h>
#include "config.h"
#include<time.h>
#define TIMES 10
/*
该函数为核心计算函数
width为输入矩阵的宽
height为输入矩阵的高
input为输入的矩阵
output为输出的矩阵
*/
static __global__ void cudaBaseline(
    const int width, //输入矩阵的宽
    const int height, //输入矩阵的高
    const float * __restrict__ input, //输入矩阵
    float* __restrict__ output //输出矩阵
    ){
    const int idy = blockIdx.y * blockDim.y + threadIdx.y; //该线程对应的行坐标
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; //该线程对应的列坐标
    if( idy < height && idx < width ){
        int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
        for(int dy = -2; dy <= 2; ++ dy ){
            int ty = idy + dy;
            if( ty >= 0 && ty < height ){
                for( int dx = -2; dx <= 2; ++ dx ){
                    int tx = idx + dx;
                    if( tx >= 0 && tx < width)
                        ++cnt[(int)input[ty * width + tx]];
                }
            }
        }
        //计算该块内元素个数, （左+中+右）*（上+中+下）
        double n = (min(2,idx) + 1 + min(2,width-idx)) * (min(2,idy) + 1 + min(2, height-idy));
        double n_div = 1/n;
        double ans = log(n); 
        for( int i = 0; i < 16; i++ ){
            if( cnt[i] )
                ans -= log((double)cnt[i]) * cnt[i] * n_div;
        }
        output[idy * width + idx] = ans;
    }
}


int main(int argc, char* argv[]){
    if( argc == 3 ){
        inputPath = argv[1];
        outputPath = argv[2];
    }
    //Open the input file
    FILE* stream = fopen(inputPath, "rb");
    if(stream == NULL){
        printf("failed to open the data file %s\n", inputPath);
        return -1;
    }
    
    //Open a stream to write out results in text
    FILE* outstream = fopen(outputPath, "wb");
    if(outstream == NULL){
        printf("failed to open the output file %s\n", outputPath);
        return -1;
    }
    
    //Read in and process the input matrix one-by-one
    int width, height, size;
    int *input;
    float *result;
    clock_t start, end;
    loadMatrix(stream, &width, &height, &input);
    size = width * height;
    printf("width = %d\n",width);
    printf("height = %d\n",height);
    printf("size = %d\n",size);
    result = (float*)malloc(sizeof(float)*size);
    
    float *input_d, *output_d;
    //在GPU上分配内存
    CHECK(hipMalloc((void**)&output_d, sizeof(float) * size));
    CHECK(hipMalloc((void**)&input_d, sizeof(float) * size));
    CHECK(hipMemcpy( input_d, input, sizeof(float) * size, hipMemcpyHostToDevice));
    
    const int BLOCK_DIM_X = 32;
    const int BLOCK_DIM_Y = 32;
    
    start = clock();
    float time = 0;
    for( int i = 0; i < TIMES; i++ ){
         dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);
         dim3 gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));
    
        cudaBaseline<<<gridDim, blockDim>>>(
            width, 
            height, 
            input_d,
            output_d);
        hipDeviceSynchronize();
        end = clock();//计时结束  
        time += (float)(end - start)/CLOCKS_PER_SEC;
    }
    //将结果写回host_result数组
    CHECK(hipMemcpy(result, output_d, sizeof(float)*size, hipMemcpyDeviceToHost));
    
    CHECK(hipFree(input_d));
    CHECK(hipFree(output_d));
    
    saveMatrix(outstream, &width, &height, &result);
    printf("Time of Cuda Baseline: %f\n",time/TIMES);
    
    free(input);
    free(result);
    input = NULL;
    result = NULL;
       
    fclose(outstream);
    fclose(stream);
    return 0;
}