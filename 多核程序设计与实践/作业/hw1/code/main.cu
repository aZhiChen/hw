#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "config.h"
#include<time.h>
//**************************************************************************
//
//  Finish your code here if need.
//
//**************************************************************************
//1维
__global__ void Matrix_add(float* A, float* B, float* C, int n){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if( tid <  n)
        C[tid] = A[tid] + B[tid];
}

int divup( int n, int m ){
    return (if(n%m)? (n/m+1):(n/m));
}

int main(int argc, char* argv[]) {
    if (argc == 3) {
        inputPath = argv[1];
        outputPath = argv[2];
    }

    // Open the input file
    FILE *stream = fopen(inputPath, "rb");
    if (stream == NULL) {
        printf("failed to open the data file %s\n", inputPath);
        return -1;
    }

    // Open a stream to write out results in text
    FILE *outStream = fopen(outputPath, "wb");
    if (outStream == NULL) {
        printf("failed to open the output file %s\n", outputPath);
        return -1;
    }

    // Read in and process the input matrix one-by-one
    int width, height, size;
    float *input1, *input2, *result;
    clock_t start,end;
    loadMatrix(stream, &width, &height, &input1);
    loadMatrix(stream, &width, &height, &input2);
    size = width * height;
    result = (float*)malloc(sizeof(float) * size);
    int block_size;
    printf("Please enter block_size!\n");
    scanf("%d\n",block_size);
    //**************************************************************************
    //
    //  Finish your code here.  Node that the array is 1D, so you should 
    //  visit the element of matrix with the way such as input[i*width+j].
    //
    //**************************************************************************
    float *input1_d, *input2_d, *result_d;
    int n_bytes = sizeof(float)*size;
    hipMalloc((void**)&input1_d, sizeof(float)*size);
    hipMalloc((void**)&input2_d, sizeof(float)*size);
    hipMalloc((void**)&result_d, sizeof(float)*size);
    start=clock();//开始计时
    hipMemcpy(input1_d, input1, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(input2_d, input2, n_bytes, hipMemcpyHostToDevice);
    Matrix_add<<<divup(size, block_size), block_size>>>(input1_d, input2_d, result_d);
    hipDeviceSynchronize();
    hipMemcpy( result, result_d, n_bytes, hipMemcpyDeviceToHost);
    end = clock();//计时结束
    hipFree(result_d);
    hipFree(input1_d);
    hipFree(input2_d);

    saveMatrix(outStream, &width, &height, &result);
    float time1=(float)(end-start)/CLOCKS_PER_SEC;
    printf("Time of GPU: %f\n", time1);
    // De-allocate the nput and the result
    free(input1);
    free(input2);
    input1 = input2 = NULL;
    free(result);
    result = NULL;
    

    // Close the output stream
    fclose(outStream);
    return 0;
}
